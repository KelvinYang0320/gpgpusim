#include <stdio.h>
#include <hip/hip_runtime.h>

#include <stdlib.h>

#define N 512

__global__ void MatAdd(float *A, float *B, float *C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // C = A + B
    if (i < N && j < N)
    {
        C[i * N + j] = A[i * N + j] + B[i * N + j];
    }
}

int main(int argc, char *argv[])
{

    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    int i;

    int x_dim, y_dim;
    char *p1;
    char *p2;
    x_dim = strtol(argv[1], &p1, 10);
    y_dim = strtol(argv[2], &p2, 10);

    // allocate the host memory
    h_A = (float *)malloc(N * N * sizeof(float));
    h_B = (float *)malloc(N * N * sizeof(float));
    h_C = (float *)malloc(N * N * sizeof(float));

    // init host data
    for (i = 0; i < (N * N); i++)
    {
        h_A[i] = 1.0;
        h_B[i] = 2.0;
        h_C[i] = 0.0;
    }

    // allocate device memory
    hipMalloc((void **)&d_A, N * N * sizeof(float));
    hipMalloc((void **)&d_B, N * N * sizeof(float));
    hipMalloc((void **)&d_C, N * N * sizeof(float));

    // transfer host data to device
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // fill in correct thread/blocks
    unsigned int x_size = x_dim;
    unsigned int y_size = y_dim;
    unsigned int grid_cols = (N + x_size - 1) / x_size;
    unsigned int grid_rows = (N + y_size - 1) / y_size;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(x_size, y_size);

    // Launch MatAdd kernel
    MatAdd<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // transfer device data back to host
    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    int all_ok = 1;
    for (i = 0; i < (N * N); i++)
    {
        if (h_C[i] != 3.0)
        {
            all_ok = 0;
            printf("wrong: %d", i);
        }
    }

    if (all_ok)
    {
        printf("all results are correct!!!\n");
    }
    else
    {
        printf("incorrect results\n");
    }

    // free memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
