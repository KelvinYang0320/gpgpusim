#include<stdio.h>
#include<hip/hip_runtime.h>


#define N 512
#define BLOCK_SIZE 32

__global__ void MatAdd(float *A, float *B, float *C){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // C = A + B (write down your codes)
    if(i < N && j < N){
        C[i*N+j] = A[i*N+j] + B[i*N+j];
    }

}

int main(){

    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    int i;

    h_A = (float*)malloc(N * N * sizeof(float));
    h_B = (float*)malloc(N * N * sizeof(float));
    h_C = (float*)malloc(N * N * sizeof(float));

    // init data
    for(i = 0; i < (N * N); i++){
	    h_A[i] = 1.0;
	    h_B[i] = 2.0;
	    h_C[i] = 0.0;
    }

    // allocate device memory
    hipMalloc((void**)&d_A,  N * N * sizeof(float));
    hipMalloc((void**)&d_B,  N * N * sizeof(float));
    hipMalloc((void**)&d_C,  N * N * sizeof(float));
    
    // transfer data to device
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // fill in correct thread/blocks
    dim3 blockSize( (N + BLOCK_SIZE - 1)/BLOCK_SIZE , (N + BLOCK_SIZE - 1)/BLOCK_SIZE);
    dim3 numBlock( BLOCK_SIZE, BLOCK_SIZE);

    // MatAdd kernel
    MatAdd<<<numBlock, blockSize>>>(d_A, d_B, d_C);
    hipDeviceSynchronize();

    // transfer data back to host
    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    for(i = 0; i < (N * N); i++){
    	
        if(h_C[i]!= 3.0){
	        printf("Error:%f, idx:%d\n", h_C[i], i);
            return 0;
	    }
    }

    printf("PASS\n");

    // free memory

    free(h_A);
    free(h_B);
    free(h_C);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}